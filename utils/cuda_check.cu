#ifndef __CUDA_CHECK_CU__
#define __CUDA_CHECK_CU__


#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

/*
 * Macro for printing error message
 */

# define CUT_CHECK_ERROR(errorMessage) do {                                 \
    hipError_t err = hipGetLastError();                                    \
    if( hipSuccess != err) {                                                \
        fprintf(stderr, "Cuda error: %s in file '%s' in line %i : %s.\n",    \
                errorMessage, __FILE__, __LINE__, hipGetErrorString( err) );\
        exit(EXIT_FAILURE);                                                  \
    }                                                                        \
    err = hipDeviceSynchronize();                                           \
    if( hipSuccess != err) {                                                \
        fprintf(stderr, "Cuda error: %s in file '%s' in line %i : %s.\n",    \
                errorMessage, __FILE__, __LINE__, hipGetErrorString( err) );\
        exit(EXIT_FAILURE);                                                  \
    } } while (0)


/*
 * Synchronizing function for checking the cuda kernel 
 */

inline void device_synchronize(hipStream_t stream)
{
        if (hipStreamSynchronize(stream) != hipSuccess) {
                CUT_CHECK_ERROR("something gone wrong\n");
        }

}

/* 
 * Macro for cuda API checking. 
 * Example: checkCuda(hipHostMalloc()); 
 */

inline hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
	if (result != hipSuccess) {
		fprintf(stderr, "CUDA Runtime Error: %sn",
		hipGetErrorString(result));
		assert(result == hipSuccess);
	}
#endif
	return result;
}


#endif

