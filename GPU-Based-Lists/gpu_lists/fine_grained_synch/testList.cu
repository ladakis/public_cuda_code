#include "hip/hip_runtime.h"
#include <stdio.h>
#include "fgsl.h"

#define NBLOCKS_TRUE 512
#define NTHREADS_TRUE 512 * 2

__global__ void kernel1(List list) {
	if(threadIdx.x == 0) {
		list.insert_node(blockIdx.x,NULL);
	
		if(blockIdx.x % 2 == 0)
			list.delete_node(blockIdx.x);
	}
}

__global__ void printList(List list) {
	//list.printlist();
}

int main() {

  int nblocks_host = 0;

  List list;
  float elapsedTime;
  hipEvent_t start, stop;

  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord( start, 0 );

  ///kernel
  kernel1<<<NBLOCKS_TRUE,NTHREADS_TRUE>>>(list);

  hipEventRecord( stop, 0 );
  hipEventSynchronize( stop );
  hipEventElapsedTime( &elapsedTime, start, stop );

  hipEventDestroy( start );
  hipEventDestroy( stop );

  printf("blockCounter1 <<< %d, %d >>> () counted %d blocks in %f ms.\n",
        NBLOCKS_TRUE,
        NTHREADS_TRUE,
        nblocks_host,
        elapsedTime);

 printList<<<1,1>>>(list);
 if(hipDeviceSynchronize()!=hipSuccess)
 	printf("Error at GPU kernel \n");
}
